#include "hip/hip_runtime.h"
#include <stdexcept>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "check.cuh"
#include "util.cuh"

__global__ void cuda_check_kernel_invocation(bool print) {
  if (print) {
    printf("GPU kernel invocation works!\n");
  }
}

bool works(bool print) {
  hipError_t err;

  cuda_check_kernel_invocation<<<1, 1>>>(print);

  err = hipPeekAtLastError();
  gpuAssert(err, __FILE__, __LINE__, false);
  if (err != hipSuccess) {
    return false;
  }

  err = hipDeviceSynchronize();
  gpuAssert(err, __FILE__, __LINE__, false);
  if (err != hipSuccess) {
    return false;
  }

  return true;
}

bool have_gpu() {
  int deviceCount = 0;
  hipError_t error;

  error = hipInit(0);
  if (error != hipSuccess) {
    return false;
  }

  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    throw std::runtime_error("Failed to query the number of CUDA devices (" +
                             std::string(std::strerror(static_cast<int>(error))) + ")\n");
  }

  return deviceCount > 0;
}
