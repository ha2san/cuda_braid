#include "hip/hip_runtime.h"
#include "braid.cuh"
#include "siphash.cuh"

__device__ void update_key(uint8_t* key, const uint8_t* buffer)
{
    for(size_t i = 0; i < FRAGMENT_BYTES; i++)
    {
        key[i] = buffer[i];
    }

    for(size_t i = 8; i < 16; i++)
    {
        key[i] = key[i-8];
    }
}

__global__ void braid(init_group_t inits, block_group_t block)
{
    for(size_t i = 0; i<N;i++)
    {
        hipMemcpy(&block[i], &inits[i & INIT_MASK], size_fragment_t, hipMemcpyDeviceToDevice);
    }
    size_t start = N - (SIZE % N);

    uint8_t buffer[FRAGMENT_BYTES];
    uint8_t key[16];

    for(size_t i = 0; i < SIZE; i++)
    {
        size_t index = (i+start) % N;
        for (size_t j = 0; j < D; j++)
        {
            size_t jump = 1 << j;
            size_t target = (index + N - jump) & INDEX_MASK;
            update_key(key,buffer);
            siphash((void*)block[target],FRAGMENT_BYTES,key,buffer,FRAGMENT_BYTES);
        }

        hipMemcpy(&block[index], &buffer, size_fragment_t, hipMemcpyDeviceToDevice);
    }
}




